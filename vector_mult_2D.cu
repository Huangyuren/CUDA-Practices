
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 64
#define cudaCheckError() {                  \
    hipError_t e = hipGetLastError();     \
    if (e != hipSuccess) {                 \
        printf("CUDA Failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                 \
    }                                       \
}

inline hipError_t cudaCheckError_inline(hipError_t result) {
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

__global__ void matrixMultiplication(int* dev_a, int* dev_b, int* dev_c, int row_a, int col_a, int col_b) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    // each (row, col) pair will match on element in resulting matrix -> with shape (row_a, col_b)
    int ret=0;
    if (row < row_a && col < col_b) {
        for(int i=0; i<col_a; ++i) {
            ret += dev_a[row * col_a + i] * dev_b[i * col_b + col_b];
        }
        dev_c[row*col_b + col] = ret;
    }
}

void matrixMultiplication_cpu(int* host_a, int* host_b, int* host_c, int row_a, int col_a, int col_b) {
    for (int i=0; i<row_a; ++i) {
        for (int j=0; j<col_b; ++j) {
            int tmp=0;
            for (int k=0; k<col_a; ++k) {
                tmp += host_a[i*col_a+k] * host_b[k*col_b+j];
            }
            host_c[i*col_b + j] = tmp;
        }
    }
}

int main(int argc, char* argv[]) {
    if(argc != 4){
        fprintf(stderr, "%s", "Usage: ./a.out $row_A $col_A $col_B $thread_count_in_block in 1Dim direction\n");
        exit(-1);
    }
	int row_a = atoi(argv[1]);
	int col_a = atoi(argv[2]);
	int col_b = atoi(argv[3]);
	int* h_a, *h_b, *h_c, *h_c_for_dev;
	cudaCheckError_inline(hipHostMalloc(&h_a, sizeof(int)*(row_a*col_a), hipHostMallocDefault));
	cudaCheckError_inline(hipHostMalloc(&h_b, sizeof(int)*(col_a*col_b), hipHostMallocDefault));
	cudaCheckError_inline(hipHostMalloc(&h_c, sizeof(int)*(row_a*col_b), hipHostMallocDefault));
    cudaCheckError_inline(hipHostMalloc(&h_c_for_dev, sizeof(int)*(row_a*col_b), hipHostMallocDefault));
    //Random initialized matrix a on host
    for(int i=0; i<row_a; ++i) {
        for(int j=0; j<col_a; ++j) {
            h_a[i*col_a+j] = rand() % 1024;
        }
    }
    //Random initialized matrix b on host
    for(int i=0; i<col_a; ++i) {
        for(int j=0; j<col_b; ++j) {
            h_a[i*col_b+j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time;
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start to count execution time of device computation
    hipEventRecord(start, 0);
	int* dev_a, *dev_b, *dev_c;
	cudaCheckError_inline(hipMalloc((void **) &dev_a, sizeof(int)*(row_a*col_a)));
	cudaCheckError_inline(hipMalloc((void **) &dev_b, sizeof(int)*(col_a*col_b)));
	cudaCheckError_inline(hipMalloc((void **) &dev_c, sizeof(int)*(row_a*col_b)));

	cudaCheckError_inline(hipMemcpy(dev_a, h_a, sizeof(int)*(row_a*col_a), hipMemcpyHostToDevice));
	cudaCheckError_inline(hipMemcpy(dev_b, h_b, sizeof(int)*(col_a*col_b), hipMemcpyHostToDevice));

    int grid_row = (row_a + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int grid_col = (col_b + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_col, grid_row);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	matrixMultiplication<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, row_a, col_a, col_b);
    cudaCheckError();
    cudaCheckError_inline(hipMemcpy(h_c_for_dev, dev_c, sizeof(int)*row_a*col_b, hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", row_a, col_a, col_a, col_b, gpu_elapsed_time_ms);
    matrixMultiplication_cpu(h_a, h_b, h_c, row_a, col_a, col_b);
    cudaCheckError_inline(hipFree(dev_a));
    cudaCheckError_inline(hipFree(dev_b));
    cudaCheckError_inline(hipFree(dev_c));
    free(h_a);
    free(h_b);
    free(h_c);
    free(h_c_for_dev);

}
