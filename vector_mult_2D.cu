
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 32
#define cudaCheckError() {                  \
    hipError_t e = hipGetLastError();     \
    if (e != hipSuccess) {                 \
        printf("CUDA Failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
        exit(EXIT_FAILURE);                 \
    }                                       \
}

inline hipError_t cudaCheckError_inline(hipError_t result) {
	if (result != hipSuccess) {
		fprintf(stderr, "CUDA error %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(result));
		assert(result == hipSuccess);
	}
	return result;
}

__global__ void matrixMultiplication(int* dev_a, int* dev_b, int* dev_c, int row_a, int col_a, int col_b) {
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    // each (row, col) pair will match on element in resulting matrix -> with shape (row_a, col_b)
    int ret=0;
    if (row < row_a && col < col_b) {
        for(int i=0; i<col_a; ++i) {
            ret += dev_a[row * col_a + i] * dev_b[i * col_b + col];
        }
        dev_c[row*col_b + col] = ret;
    }
}

void matrixMultiplication_cpu(int* host_a, int* host_b, int* host_c, int row_a, int col_a, int col_b) {
    for (int i=0; i<row_a; ++i) {
        for (int j=0; j<col_b; ++j) {
            int tmp=0;
            for (int k=0; k<col_a; ++k) {
                tmp += host_a[i*col_a+k] * host_b[k*col_b+j];
            }
            host_c[i*col_b + j] = tmp;
        }
    }
}
bool verifyResult(int* h_c, int* h_c_result, int rows, int cols) {
    for(int i=0; i<rows; ++i) {
        for(int j=0; j<cols; ++j) {
            if(h_c[i*cols + j] != h_c_result[i*cols + j]){
                printf("Host: %d, Device: %d\n", h_c[i*cols + j], h_c_result[i*cols + j]);
                return false;
            }
        }
    }
    return true;
}

int main(int argc, char* argv[]) {
    if(argc != 4){
        fprintf(stderr, "%s", "Usage: ./a.out $row_A $col_A $col_B $thread_count_in_block in 1Dim direction\n");
        exit(-1);
    }
	int row_a = atoi(argv[1]);
	int col_a = atoi(argv[2]);
	int col_b = atoi(argv[3]);
	int* h_a, *h_b, *h_c, *h_c_result;
	cudaCheckError_inline(hipHostMalloc((void **) &h_a, sizeof(int)*(row_a*col_a), hipHostMallocDefault));
	cudaCheckError_inline(hipHostMalloc((void **) &h_b, sizeof(int)*(col_a*col_b), hipHostMallocDefault));
	cudaCheckError_inline(hipHostMalloc((void **) &h_c, sizeof(int)*(row_a*col_b), hipHostMallocDefault));
    cudaCheckError_inline(hipHostMalloc((void **) &h_c_result, sizeof(int)*(row_a*col_b), hipHostMallocDefault));
    //Random initialized matrix a on host
    for(int i=0; i<row_a; ++i) {
        for(int j=0; j<col_a; ++j) {
            h_a[i*col_a+j] = rand() % 1024;
        }
    }
    //Random initialized matrix b on host
    for(int i=0; i<col_a; ++i) {
        for(int j=0; j<col_b; ++j) {
            h_a[i*col_b+j] = rand() % 1024;
        }
    }

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;
	hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    // Start counting execution time of device computation
    hipEventRecord(start, 0);
	int* dev_a, *dev_b, *dev_c;
	cudaCheckError_inline(hipMalloc((void **) &dev_a, sizeof(int)*(row_a*col_a)));
	cudaCheckError_inline(hipMalloc((void **) &dev_b, sizeof(int)*(col_a*col_b)));
	cudaCheckError_inline(hipMalloc((void **) &dev_c, sizeof(int)*(row_a*col_b)));

	cudaCheckError_inline(hipMemcpy(dev_a, h_a, sizeof(int)*(row_a*col_a), hipMemcpyHostToDevice));
	cudaCheckError_inline(hipMemcpy(dev_b, h_b, sizeof(int)*(col_a*col_b), hipMemcpyHostToDevice));

    int grid_row = (row_a + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int grid_col = (col_b + BLOCK_SIZE - 1) / BLOCK_SIZE;
	dim3 dimGrid(grid_col, grid_row);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	matrixMultiplication<<<dimGrid, dimBlock>>>(dev_a, dev_b, dev_c, row_a, col_a, col_b);
    cudaCheckError_inline(hipDeviceSynchronize());
    cudaCheckError_inline(hipMemcpy(h_c_result, dev_c, sizeof(int)*row_a*col_b, hipMemcpyDeviceToHost));
    
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n", row_a, col_a, col_a, col_b, gpu_elapsed_time_ms);

    //Start counting execution time of cpu computation
    hipEventRecord(start, 0);
    matrixMultiplication_cpu(h_a, h_b, h_c, row_a, col_a, col_b);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n", row_a, col_a, col_a, col_b, cpu_elapsed_time_ms);
    bool check = verifyResult(h_c, h_c_result, row_a, col_b);
    if(!check) {
        fprintf(stderr, "Error, result not matched.\n");
        exit(4);
    }else{
        printf("Congratulations, results match !!\n");
    }
    float speedups = cpu_elapsed_time_ms / gpu_elapsed_time_ms;
    printf("Overall speedup = %f\n", speedups);

    cudaCheckError_inline(hipFree(dev_a));
    cudaCheckError_inline(hipFree(dev_b));
    cudaCheckError_inline(hipFree(dev_c));

    cudaCheckError_inline(hipHostFree(h_a));
    cudaCheckError_inline(hipHostFree(h_b));
    cudaCheckError_inline(hipHostFree(h_c));
    cudaCheckError_inline(hipHostFree(h_c_result));

}
